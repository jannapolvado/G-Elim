/******************************************************************************
 * Janna Polvado
 * Massive Parallel Programming
 * Gaussian Elimination 
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"

int main (int argc, char *argv[])
{

    //printf("THIS IS A TEST!!!!!\n");
    Timer timer, runtimer;
    hipError_t cuda_ret;
    //startTime(&runtimer);
    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

   
    unsigned int matArow, matAcol;
    dim3 dim_grid, dim_block;
    float *A_h, *A_d, *result, sum, rval;
    //float *B_h, *B_d;///////////////////////////////////
    int numvar;
    size_t A_sz;//, B_sz;/////////////////////////
    
   

   
    if (argc == 1) {
        matArow = 1000;
        numvar = matArow;
        matAcol = matArow+1;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        numvar = matArow;
        matAcol = matArow+1;
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./g-elim                # All matrices are 1000 x 1000"
      "\n    Usage: ./g-elim <m>            # All matrices are m x m"
     
      "\n");
        exit(0);
    }

    
    A_sz = matArow*matAcol;
    //B_sz = A_sz;///////////////////////////////////

    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i = 0; i < matArow; i++) { 
        for(unsigned int j = 0; j < matAcol; j++){
        
            A_h[i*matAcol+j] = (rand()%100);
            //printf("%.2f   ", A_h[i*matAcol+j]);
        }
        //printf("\n");
    }
    //printf("\n\n");
        
    //B_h = (float*)malloc(sizeof(float)*B_sz);////////////////////
    //for(unsigned int i = 0; i<B_sz; i++){
        //B_h[i] = 1;
        //printf("%.2f ", B_h[i]);
    //}////////////////////////////////////////////
    //printf("\n");
    
    result = (float*)malloc(sizeof(float)*numvar);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n", matArow, matAcol,
        matArow, matAcol);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    cuda_ret = hipMalloc((void**) &A_d, A_sz*sizeof(float));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    //cuda_ret = hipMalloc((void**) &B_d, B_sz*sizeof(float));///////////////////////////
    //if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");//////////////

  

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMemcpy(A_d,A_h,A_sz*sizeof(float),hipMemcpyHostToDevice);
  
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    
    // Launch kernel using standard Gelim interface ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    
    //basicGelim(A_d,B_d,numvar);//////////////////////////
    basicGelim(A_d,numvar);

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    printf("Copying data from device to host...\n"); fflush(stdout);
    startTime(&timer);

    hipMemcpy(A_h,A_d,A_sz*sizeof(float),hipMemcpyDeviceToHost);//should be all b////////
    
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Prints U matrix

    /*for(int i = 0; i < numvar; i++){
        for(int j = 0; j< numvar+1; j++){
            printf("%.2f    ", A_h[i*(numvar+1)+j]);///////////////Ah with Bh
        }
        printf("\n");
    }*/////////////////////////////////////comment out section  again


    printf("Backwards Substitution..."); fflush(stdout);
    startTime(&timer);

    // BACKWARDS SUBSTITUTION
    //result = (float*)malloc(sizeof(float)*numvar);
    for(int i = 0; i < numvar; i++){
        result[i] = 1.0;
    }

    for(int i = numvar-1; i >= 0; i--){
        sum = 0;
        int j;
        for(j = numvar-1; j > i; j--){
            sum += result[j]*A_h[i*(numvar+1) + j];///////////////////b instead of a
        }
        rval = A_h[i*(numvar+1) + numvar] - sum;////////////instead
        result[i] = rval/A_h[i*(numvar+1)+j];////////////////instead
       // printf("%.2f\n", resulti[i]);
    }

    stopTime(&timer);
    printf("%f s\n", elapsedTime(timer));
    // Prints solution vector
    /*printf("Printing Results...\n");
    for(int i =0; i < numvar; i++){
        printf("%.6f\n", result[i]);
    }*/


    // Verify correctness -----------------------------------------------------

    printf("Verifying results...\n"); fflush(stdout);

    verify(A_h, result, numvar);///////////////////////////fix this


    // Free memory ------------------------------------------------------------

    free(A_h);
    //free(B_h);/////////////////////////////////////
   

    //INSERT CODE HERE
    hipFree(A_d);
    //hipFree(B_d);/////////////////////////
    
    //stopTime(&runtimer);
    //printf("%f s\n", elapsedTime(runtimer));
    return 0;

}
