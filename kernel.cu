/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SZ 16 

__global__ void Gelim(float *A,  int numvar){/////////////add b back in middle

    __shared__ float A_s[TILE_SZ][TILE_SZ];
    int Tirow = threadIdx.y;
    int Ticol = threadIdx.x;
  
    A_s[Tirow][Ticol] = A[(Tirow * (numvar +1)) + Ticol];
       
    for(int i = 1; i < numvar; i++){
       
        if((Tirow +i) < (numvar)){
          
            float multiplier = A_s[Tirow+i][i-1]/A_s[i-1][i-1];
            if(Tirow  <= Ticol+1){
                A_s[Tirow+i][Ticol] -= (multiplier * A_s[i-1][Ticol]);
            }
            else{
                A_s[Tirow+i][Ticol] = 0;
            }
            __syncthreads();
        }
        //__syncthreads();
    }
    A[Tirow *(numvar+1) +Ticol] = A_s[Tirow][Ticol];/////////////replace a with b

}

   

void basicGelim(float *A,  int numvar){ ////////add b back in middle
    dim3 block(numvar+1, numvar, 1);
    dim3 grid(1,1,1);

    Gelim<<<grid,block>>>(A,numvar);//////////b  back in mid

}

/*void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }*/

    // Initialize thread block and kernel grid dimensions ---------------------

   /* const unsigned int BLOCK_SIZE = TILE_SZ;
    dim3 block(BLOCK_SIZE,BLOCK_SIZE);
    dim3 grid((n+BLOCK_SIZE-1)/BLOCK_SIZE,(m+BLOCK_SIZE-1)/BLOCK_SIZE);
    //INSERT CODE HERE

    // Invoke CUDA kernel -----------------------------------------------------
    mysgemm<<<grid,block>>>(m,n,k,A,B,C);
    //INSERT CODE HERE
    dim3 block(numvar+1,numvar,1);
    dim3 grid(1,1,1);



}*/
